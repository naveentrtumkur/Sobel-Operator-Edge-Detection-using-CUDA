#include "hip/hip_runtime.h"
	/***
	 * File Name: sobel_edge.cu
	 * Description: This Program Performs Sobel edge detection operations on a .bmp, once by a
	 * serial algorithm, and once by a massively parallel CUDA algorithm.
	 */
	
	#include <stdio.h>
	#include <math.h>
	#include <stdlib.h>
	#include <stdio.h>
	#include <time.h>
	
	//C version of Code to be specified as extern,
	//because CUDA uses g++ version of compiler.
	extern "C"
	{
	#include "read_bmp.h"
	}
	
	//Define white,black and threshold values.
	#define PIXEL_BLACK 0
	#define PIXEL_WHITE 255
	#define PERCENT_BLACK_THRESHOLD 0.75
	
	//Define the thread hierarchy being used.
	#define CUDA_GRIDS 10
	#define CUDA_BLOCKS_PER_GRID 32
	#define CUDA_THREADS_PER_BLOCK 1024
	
	#define NS_PER_SEC 1000000000

	//Returns the pixel value when we pass row,coulmn value
	#define getPixelValue(row, col, dim) \
	   (((row) * (dim)) + (col))
	
	//Values required for timer calculation.
	static struct timespec serialStart;
	static struct timespec serialEnd;
	static struct timespec parallelStart;
	static struct timespec parallelEnd;

	/*
	 * Function to Display all the required information: matrix and CUDA parameters.
	 *
	 * @param inputFile -- name of the input image
	 * @param serialOutputFile -- name of the serial output image
	 * @param parallelOutputFile -- name of the parallel output image
	 * @param imageHeight -- Height of the image in pixels
	 * @param imageWidth -- Width of the image in pixels
	 */
	void DisplayParameters(char *inputFile, char *serialOutputFile, char *cudaOutputFile,
	   			int imageHeight,int imageWidth)
	{
	   printf("********************************************************************************\n");
	   printf("Serial and  CUDA Sobel edge detection.\n");
	   printf("\n");
	   printf("Input image: %s \t(Height is: %d pixels, Width is : %d pixels)\n", inputFile, imageHeight, imageWidth);
	   printf("Serial output image is: \t%s\n", serialOutputFile);
	   printf("CUDA output image is: \t%s\n", cudaOutputFile);
	   printf("\n");
	   printf("CUDA computation specifications are:\n");
	   printf("Grids =  %d grids\n", CUDA_GRIDS);
	   printf("Blocks=  %d blocks\n", CUDA_BLOCKS_PER_GRID);
	   printf("tpb= %d threads per block\n", CUDA_THREADS_PER_BLOCK);
	   printf("\n");
	   printf("*********************************************************************************\n");
	}
	

	/*
	 * Function to display information of threshold,timing and convergence results onto the screen.
	 *
	 * @param serialConvergenceThreshold
	 * @param serialConvergenceThreshold
	 */
	void DisplayResults(int serialConvergenceThreshold, int parallelConvergenceThreshold)
	{
	   printf("*******************************************************************************\n");
	   printf("Time taken for serial Sobel edge detection: %lf\n",
	      (getPixelValue(serialEnd.tv_sec, serialEnd.tv_nsec, NS_PER_SEC)
	      - getPixelValue(serialStart.tv_sec, serialStart.tv_nsec, NS_PER_SEC))
	      / ((double)NS_PER_SEC));

	   printf("Convergence Threshold: %d\n", serialConvergenceThreshold);
	   printf("\n");
	

	   printf("Time taken for CUDA Parallel Sobel edge detection: %lf\n",
	      (getPixelValue(parallelEnd.tv_sec, parallelEnd.tv_nsec, NS_PER_SEC)
	      - getPixelValue(parallelStart.tv_sec, parallelStart.tv_nsec, NS_PER_SEC))
	      / ((double)NS_PER_SEC));
	

	   printf("Convergence Threshold: %d\n", parallelConvergenceThreshold);
	   printf("********************************************************************************\n");
	  
	}
	

	/*
	 * Serial algorithm to perform Sobel edge detection on an input pixel
	 * image which is at different brightness thresholds until a certain percentage of
	 * pixels in the output pixel buffer are black(75% in our case).
	 *
	 * @param input -- input pixel buffer
	 * @param output -- output pixel buffer
	 * @param height -- height of pixel image
	 * @param width -- width of pixel image
	 * @return -- gradient threshold at which PERCENT_BLACK_THRESHOLD(75%) of pixels are black
	 */
	int SerialSobelEdgeDetection(uint8_t *input, uint8_t *output, int height, int width)
	{
	    //printf("height=%d, width =%d\n",height,width);
	    int i=0,j=0;
	    int gradientThreshold=0, blackPixelCount = 0 , boundary = height * width * 3 / 4;
	    int whitePixelCount =0;
	    //printf("value 75% = %d",boundary);
	    // Loop until blackpixel counts are less than boundary(75% black cells)
	    while(blackPixelCount < boundary)
	    {
	        gradientThreshold = gradientThreshold +1;
	        //printf("%d -- blackpix\n",blackPixelCount);	 
	        //printf("%d -- gradthresh\n",gradientThreshold);	 
	        blackPixelCount = 0;
	        for(i=1;i<(height-1);i++)
		{
		    for(j=1; j<(width-1);j++)
			{
		
		 	    double Gx = (1*input[getPixelValue(i - 1, j + 1, width)])
         				+ (-1 * input[getPixelValue(i-1, j - 1, width)])
         				+ (2 * input[getPixelValue(i, j + 1, width)])
         				+ (-2 * input[getPixelValue(i, j - 1, width)])
         				+ (1 * input[getPixelValue(i + 1, j + 1, width)])
         				+ (-1 * input[getPixelValue(i + 1, j - 1, width)]);

      			    double Gy = (1* input[getPixelValue(i - 1, j - 1, width)])
         		    + (2 * input[getPixelValue(i - 1, j, width)])
         		    + (1 * input[getPixelValue(i - 1, j + 1, width)])
        		    + (-1 * input[getPixelValue(i + 1, j - 1, width)])
        		    + (-2 * input[getPixelValue(i + 1, j, width)])
       			    + (-1 * input[getPixelValue(i + 1, j + 1, width)]);
			   
			   //Instead of squareroot, square threshold and compare directly with magnitude value 
      			   if(((Gx * Gx) + (Gy * Gy)) > (gradientThreshold * gradientThreshold))
      			   {
				//set the output value to white
         			output[getPixelValue(i,j,width)] = PIXEL_WHITE;
				whitePixelCount++;
     			   }
      			   else
      			   {
				//set the output value to black
         			output[getPixelValue(i,j,width)] = PIXEL_BLACK;
				blackPixelCount++;
      			   }
		        }
		    }
		    //printf("white=%d",whitePixelCount);
		    //printf("blck=%d",blackPixelCount);

	    }
	   //printf("%d -- gradthresh\n",gradientThreshold);	 
	   return gradientThreshold;
	}


	/*
	 * This is Parallel CUDA kernel function that performs a Sobel edge detection
	 * on a group of pixels. This kernel function is called from host's function.
	 *
	 */
	__global__ void CudaSobelEdgeDetection(uint8_t *input, uint8_t *output, int height, int width, int gradientThreshold)
	{
	   int row = 0;
	   for(int i = 0; row < (height - 1); i++)
	   {
	      // Let the blockIdx increment beyond its dimension for cyclic distribution of the test pixels
	      int blockRow = (i * gridDim.x) + blockIdx.x;
	

	      // Calculate the row/col in the image buffer that this thread is on
	      row = (getPixelValue(blockRow, threadIdx.x, blockDim.x) / (width - 2)) + 1;
	      int col = (getPixelValue(blockRow, threadIdx.x, blockDim.x) % (width - 2)) + 1;
	

	      // Calculate Sobel magnitude of gradient directly, instead of using Sobel_Magnitude utility
	      double Gx = // ( * input[getPixelValue(row - 1, col - 1, width)])
	         	+ ( +1 * input[getPixelValue(row - 1, col + 1, width)])
	         	+ ( -1 * input[getPixelValue(row - 1, col - 1, width)])
	         	+ ( +2 * input[getPixelValue(row, col + 1, width)])
	        	+ ( -2 * input[getPixelValue(row, col - 1, width)])
	         	+ ( +1  * input[getPixelValue(row + 1, col + 1, width)])
	         	+ ( -1  * input[getPixelValue(row + 1, col - 1, width)]);
	

	      double Gy = ( +1 * input[getPixelValue(row - 1, col - 1, width)])
	         	+ ( +2 * input[getPixelValue(row - 1, col, width)])
	        	+ ( +1 * input[getPixelValue(row - 1, col + 1, width)])
	         	+ ( -1 * input[getPixelValue(row + 1, col - 1, width)])
	         	+ ( -2 * input[getPixelValue(row + 1, col, width)])
	         	+ ( -1 * input[getPixelValue(row + 1, col + 1, width)]);

	      if(((Gx * Gx) + (Gy * Gy)) > (gradientThreshold * gradientThreshold))
	      {
		 //set the output value to white.
	         output[getPixelValue(row, col, width)] = PIXEL_WHITE;
	      }
	      else
	      {
		 //set the output pixel value to black.
	         output[getPixelValue(row, col, width)] = PIXEL_BLACK;
	      }
	   }
	}
	

	/*
	 * Parallel algorithm to perform a Sobel edge detection on an input pixel
	 * image at different brightness thresholds until a certain percentage of
	 * pixels in the output pixel buffer are black.
	 *
	 * @param input -- input pixel buffer
	 * @param output -- output pixel buffer
	 * @param height -- height of pixel image
	 * @param width -- width of pixel image
	 * @return -- gradient threshold at which PERCENT_BLACK_THRESHOLD(75%) of pixels are black
	 */
	__host__ int ParallelSobelEdgeDetection(uint8_t *input, uint8_t *output, int height, int width)
	{
	   int numBlocks = CUDA_BLOCKS_PER_GRID;  
	   int threadsPerBlock = CUDA_THREADS_PER_BLOCK;
	   size_t imageMemSize =  height * width * sizeof(uint8_t);
	   uint8_t *deviceInputImage, *deviceOutputImage;
	
	   // Allocate device memory
	   hipMalloc((void **)&deviceInputImage, imageMemSize);
	   hipMalloc((void **)&deviceOutputImage, imageMemSize);

	   // Copy host input image to device
	   hipMemcpy(deviceInputImage, input, imageMemSize, hipMemcpyHostToDevice);
	
	   //define the device data-structures	
	   dim3 dimGrid(numBlocks);
	   dim3 dimBlock(threadsPerBlock);
	
	   //Perform Parallel Cuda Sobel edge detetction by calling the kernel.
	   int gradientThreshold, blackPixelCount = 0;
	   for(gradientThreshold = 0; blackPixelCount < (height * width * 75 / 100); gradientThreshold++)
	   {
	      // Launching the Kernel
	      CudaSobelEdgeDetection<<<dimGrid, dimBlock>>>(deviceInputImage, deviceOutputImage, height, width, gradientThreshold);

	      // Copy the device results array back to host
	      hipMemcpy(output, deviceOutputImage, imageMemSize, hipMemcpyDeviceToHost);

	      // Count the number of black pixels
	      blackPixelCount = 0;
	      for(int row = 1; row < (height - 1); row++)
	      {
	         for(int col = 1; col < (width - 1); col++)
	         {
	            if(output[getPixelValue(row, col, width)] == PIXEL_BLACK)
	            {
	               blackPixelCount++;
	            }
	         }
	      }
	   }
	   //Due to for loop increment, we need to decrement value and return it.
	   return gradientThreshold-1;
	}


	/*
	* Main function.
	*/
	int main(int argc, char* argv[])
	{
	   // Check for correct number of comand line args
	   if (argc != 4)
	   {
	      printf("Error:Incorrect arguments: <input_img.bmp> <serial_output_img.bmp> <cuda_output_img.bmp> Please try again..\n");
	      return 0;
	   }

	   // Open the files specified by the command line args
	   FILE *inputFile = fopen(argv[1], "rb");
	   FILE *serialOutputFile = fopen(argv[2], "wb");
	   FILE *cudaOutputFile = fopen(argv[3], "wb");

	   //check if the file is valid and can be opened.
	   if(inputFile == NULL)
	   {
	      printf("Error: %s file could not be opened for reading.", argv[1]);
	   }

	   // Read in input image and allocate space for new output image buffers
	   uint8_t *inputImage = (uint8_t *)read_bmp_file(inputFile);
	   uint8_t *serialOutputImage = (uint8_t *)malloc(get_num_pixel());
	   uint8_t *cudaOutputImage = (uint8_t *)malloc(get_num_pixel());

	   // Display GPU related information by calling this function.
	   DisplayParameters(argv[1], argv[2], argv[3], get_image_height(), get_image_width());
	
	   // Call the serial function for serial sobel edge detection.
	   printf("Performing serial Sobel edge detection.\n");
	   clock_gettime(CLOCK_REALTIME, &serialStart);
	   int serialConvergenceThreshold = SerialSobelEdgeDetection(inputImage, serialOutputImage, get_image_height(), get_image_width());
	   clock_gettime(CLOCK_REALTIME, &serialEnd);
	
	   // Call the CUDA function for Parallel sobel edge detection
	   printf("Performing CUDA parallel Sobel edge detection.\n");
	   clock_gettime(CLOCK_REALTIME, &parallelStart);
	   int parallelConvergenceThreshold = ParallelSobelEdgeDetection(inputImage, cudaOutputImage, get_image_height(), get_image_width());
	   clock_gettime(CLOCK_REALTIME, &parallelEnd);
	

	   //DisplayResults for parallel and serial ConvergenceThreshold.
	   DisplayResults(serialConvergenceThreshold,parallelConvergenceThreshold);

	   // Write output image buffers. Closes files and frees buffers.
	   write_bmp_file(serialOutputFile, serialOutputImage);
	   write_bmp_file(cudaOutputFile, cudaOutputImage);
	   
	   return 0;	
	}

